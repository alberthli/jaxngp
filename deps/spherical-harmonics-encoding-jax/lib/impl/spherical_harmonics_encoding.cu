#include "hip/hip_runtime.h"
#ifdef __HIPCC__
#define FE_INLINE __host__ __device__
#else
#define FE_INLINE inline
#endif

#include <cstdint>

#include "spherical_harmonics_encoding.h"
#include "../serde.h"


namespace shjax {

namespace {

__inline__ void check_throw(hipError_t error) {
    if (error != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(error));
    }
}

// debugging kernel for inspecting data passed to custom op
__global__ void copy_left_to_right(std::uint32_t length, float const *lhs, float * const rhs) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    #pragma unroll
    for (int i = index; i < length; i += stride) {
        rhs[i] = lhs[i];
    }
}

template <typename real_t>
__device__ __inline__ void sh_enc(
    std::uint32_t const degree,
    float const x,
    float const y,
    float const z,
    real_t * const __restrict__ o
) {
    // adapted from <https://github.com/NVlabs/tiny-cuda-nn/blob/39df2387a684e4fe0cfa33542aebf5eab237716b/include/tiny-cuda-nn/encodings/spherical_harmonics.h#L52-L123>

    float xy=x*y, xz=x*z, yz=y*z, x2=x*x, y2=y*y, z2=z*z;
    float x4=x2*x2, y4=y2*y2, z4=z2*z2;
    float x6=x4*x2, y6=y4*y2, z6=z4*z2;

    o[0] = (real_t)(0.28209479177387814f);                          // 1/(2*sqrt(pi))
    if (degree <= 1) { return; }

    o[1] = (real_t)(-0.48860251190291987f*y);                               // -sqrt(3)*y/(2*sqrt(pi))
    o[2] = (real_t)(0.48860251190291987f*z);                                // sqrt(3)*z/(2*sqrt(pi))
    o[3] = (real_t)(-0.48860251190291987f*x);                               // -sqrt(3)*x/(2*sqrt(pi))
    if (degree <= 2) { return; }

    o[4] = (real_t)(1.0925484305920792f*xy);                                // sqrt(15)*xy/(2*sqrt(pi))
    o[5] = (real_t)(-1.0925484305920792f*yz);                               // -sqrt(15)*yz/(2*sqrt(pi))
    o[6] = (real_t)(0.94617469575755997f*z2 - 0.31539156525251999f);                         // sqrt(5)*(3*z2 - 1)/(4*sqrt(pi))
    o[7] = (real_t)(-1.0925484305920792f*xz);                               // -sqrt(15)*xz/(2*sqrt(pi))
    o[8] = (real_t)(0.54627421529603959f*x2 - 0.54627421529603959f*y2);                              // sqrt(15)*(x2 - y2)/(4*sqrt(pi))
    if (degree <= 3) { return; }

    o[9] = (real_t)(0.59004358992664352f*y*(-3.0f*x2 + y2));                         // sqrt(70)*y*(-3*x2 + y2)/(8*sqrt(pi))
    o[10] = (real_t)(2.8906114426405538f*xy*z);                             // sqrt(105)*xy*z/(2*sqrt(pi))
    o[11] = (real_t)(0.45704579946446572f*y*(1.0f - 5.0f*z2));                                // sqrt(42)*y*(1 - 5*z2)/(8*sqrt(pi))
    o[12] = (real_t)(0.3731763325901154f*z*(5.0f*z2 - 3.0f));                         // sqrt(7)*z*(5*z2 - 3)/(4*sqrt(pi))
    o[13] = (real_t)(0.45704579946446572f*x*(1.0f - 5.0f*z2));                                // sqrt(42)*x*(1 - 5*z2)/(8*sqrt(pi))
    o[14] = (real_t)(1.4453057213202769f*z*(x2 - y2));                              // sqrt(105)*z*(x2 - y2)/(4*sqrt(pi))
    o[15] = (real_t)(0.59004358992664352f*x*(-x2 + 3.0f*y2));                                // sqrt(70)*x*(-x2 + 3*y2)/(8*sqrt(pi))
    if (degree <= 4) { return; }

    o[16] = (real_t)(2.5033429417967046f*xy*(x2 - y2));                             // 3*sqrt(35)*xy*(x2 - y2)/(4*sqrt(pi))
    o[17] = (real_t)(1.7701307697799304f*yz*(-3.0f*x2 + y2));                                // 3*sqrt(70)*yz*(-3*x2 + y2)/(8*sqrt(pi))
    o[18] = (real_t)(0.94617469575756008f*xy*(7.0f*z2 - 1.0f));                               // 3*sqrt(5)*xy*(7*z2 - 1)/(4*sqrt(pi))
    o[19] = (real_t)(0.66904654355728921f*yz*(3.0f - 7.0f*z2));                               // 3*sqrt(10)*yz*(3 - 7*z2)/(8*sqrt(pi))
    o[20] = (real_t)(-3.1735664074561294f*z2 + 3.7024941420321507f*z4 + 0.31735664074561293f);                                // 3*(-30*z2 + 35*z4 + 3)/(16*sqrt(pi))
    o[21] = (real_t)(0.66904654355728921f*xz*(3.0f - 7.0f*z2));                               // 3*sqrt(10)*xz*(3 - 7*z2)/(8*sqrt(pi))
    o[22] = (real_t)(0.47308734787878004f*(x2 - y2)*(7.0f*z2 - 1.0f));                                // 3*sqrt(5)*(x2 - y2)*(7*z2 - 1)/(8*sqrt(pi))
    o[23] = (real_t)(1.7701307697799304f*xz*(-x2 + 3.0f*y2));                                // 3*sqrt(70)*xz*(-x2 + 3*y2)/(8*sqrt(pi))
    o[24] = (real_t)(-3.7550144126950569f*x2*y2 + 0.62583573544917614f*x4 + 0.62583573544917614f*y4);                         // 3*sqrt(35)*(-6*x2*y2 + x4 + y4)/(16*sqrt(pi))
    if (degree <= 5) { return; }

    o[25] = (real_t)(0.65638205684017015f*y*(10.0f*x2*y2 - 5.0f*x4 - y4));                            // 3*sqrt(154)*y*(10*x2*y2 - 5*x4 - y4)/(32*sqrt(pi))
    o[26] = (real_t)(8.3026492595241645f*xy*z*(x2 - y2));                           // 3*sqrt(385)*xy*z*(x2 - y2)/(4*sqrt(pi))
    o[27] = (real_t)(-0.48923829943525038f*y*(3.0f*x2 - y2)*(9.0f*z2 - 1.0f));                         // -sqrt(770)*y*(3*x2 - y2)*(9*z2 - 1)/(32*sqrt(pi))
    o[28] = (real_t)(4.7935367849733241f*xy*z*(3.0f*z2 - 1.0f));                              // sqrt(1155)*xy*z*(3*z2 - 1)/(4*sqrt(pi))
    o[29] = (real_t)(0.45294665119569694f*y*(14.0f*z2 - 21.0f*z4 - 1.0f));                             // sqrt(165)*y*(14*z2 - 21*z4 - 1)/(16*sqrt(pi))
    o[30] = (real_t)(0.1169503224534236f*z*(-70.0f*z2 + 63.0f*z4 + 15.0f));                            // sqrt(11)*z*(-70*z2 + 63*z4 + 15)/(16*sqrt(pi))
    o[31] = (real_t)(0.45294665119569694f*x*(14.0f*z2 - 21.0f*z4 - 1.0f));                             // sqrt(165)*x*(14*z2 - 21*z4 - 1)/(16*sqrt(pi))
    o[32] = (real_t)(2.3967683924866621f*z*(x2 - y2)*(3.0f*z2 - 1.0f));                               // sqrt(1155)*z*(x2 - y2)*(3*z2 - 1)/(8*sqrt(pi))
    o[33] = (real_t)(-0.48923829943525038f*x*(x2 - 3.0f*y2)*(9.0f*z2 - 1.0f));                         // -sqrt(770)*x*(x2 - 3*y2)*(9*z2 - 1)/(32*sqrt(pi))
    o[34] = (real_t)(2.0756623148810411f*z*(-6.0f*x2*y2 + x4 + y4));                         // 3*sqrt(385)*z*(-6*x2*y2 + x4 + y4)/(16*sqrt(pi))
    o[35] = (real_t)(0.65638205684017015f*x*(10.0f*x2*y2 - x4 - 5.0f*y4));                            // 3*sqrt(154)*x*(10*x2*y2 - x4 - 5*y4)/(32*sqrt(pi))
    if (degree <= 6) { return; }

    o[36] = (real_t)(1.3663682103838286f*xy*(-10.0f*x2*y2 + 3.0f*x4 + 3.0f*y4));                               // sqrt(6006)*xy*(-10*x2*y2 + 3*x4 + 3*y4)/(32*sqrt(pi))
    o[37] = (real_t)(2.3666191622317521f*yz*(10.0f*x2*y2 - 5.0f*x4 - y4));                            // 3*sqrt(2002)*yz*(10*x2*y2 - 5*x4 - y4)/(32*sqrt(pi))
    o[38] = (real_t)(2.0182596029148963f*xy*(x2 - y2)*(11.0f*z2 - 1.0f));                             // 3*sqrt(91)*xy*(x2 - y2)*(11*z2 - 1)/(8*sqrt(pi))
    o[39] = (real_t)(-0.92120525951492349f*yz*(3.0f*x2 - y2)*(11.0f*z2 - 3.0f));                               // -sqrt(2730)*yz*(3*x2 - y2)*(11*z2 - 3)/(32*sqrt(pi))
    o[40] = (real_t)(0.92120525951492349f*xy*(-18.0f*z2 + 33.0f*z4 + 1.0f));                           // sqrt(2730)*xy*(-18*z2 + 33*z4 + 1)/(32*sqrt(pi))
    o[41] = (real_t)(0.58262136251873131f*yz*(30.0f*z2 - 33.0f*z4 - 5.0f));                            // sqrt(273)*yz*(30*z2 - 33*z4 - 5)/(16*sqrt(pi))
    o[42] = (real_t)(6.6747662381009842f*z2 - 20.024298714302954f*z4 + 14.684485723822165f*z6 - 0.31784601133814211f);                         // sqrt(13)*(105*z2 - 315*z4 + 231*z6 - 5)/(32*sqrt(pi))
    o[43] = (real_t)(0.58262136251873131f*xz*(30.0f*z2 - 33.0f*z4 - 5.0f));                            // sqrt(273)*xz*(30*z2 - 33*z4 - 5)/(16*sqrt(pi))
    o[44] = (real_t)(0.46060262975746175f*(x2 - y2)*(11.0f*z2*(3.0f*z2 - 1.0f) - 7.0f*z2 + 1.0f));                               // sqrt(2730)*(x2 - y2)*(11*z2*(3*z2 - 1) - 7*z2 + 1)/(64*sqrt(pi))
    o[45] = (real_t)(-0.92120525951492349f*xz*(x2 - 3.0f*y2)*(11.0f*z2 - 3.0f));                               // -sqrt(2730)*xz*(x2 - 3*y2)*(11*z2 - 3)/(32*sqrt(pi))
    o[46] = (real_t)(0.50456490072872406f*(11.0f*z2 - 1.0f)*(-6.0f*x2*y2 + x4 + y4));                          // 3*sqrt(91)*(11*z2 - 1)*(-6*x2*y2 + x4 + y4)/(32*sqrt(pi))
    o[47] = (real_t)(2.3666191622317521f*xz*(10.0f*x2*y2 - x4 - 5.0f*y4));                            // 3*sqrt(2002)*xz*(10*x2*y2 - x4 - 5*y4)/(32*sqrt(pi))
    o[48] = (real_t)(10.247761577878714f*x2*y4 - 10.247761577878714f*x4*y2 + 0.6831841051919143f*x6 - 0.6831841051919143f*y6);                         // sqrt(6006)*(15*x2*y4 - 15*x4*y2 + x6 - y6)/(64*sqrt(pi))
    if (degree <= 7) { return; }

    o[49] = (real_t)(0.70716273252459627f*y*(-21.0f*x2*y4 + 35.0f*x4*y2 - 7.0f*x6 + y6));                              // 3*sqrt(715)*y*(-21*x2*y4 + 35*x4*y2 - 7*x6 + y6)/(64*sqrt(pi))
    o[50] = (real_t)(5.2919213236038001f*xy*z*(-10.0f*x2*y2 + 3.0f*x4 + 3.0f*y4));                             // 3*sqrt(10010)*xy*z*(-10*x2*y2 + 3*x4 + 3*y4)/(32*sqrt(pi))
    o[51] = (real_t)(-0.51891557872026028f*y*(13.0f*z2 - 1.0f)*(-10.0f*x2*y2 + 5.0f*x4 + y4));                          // -3*sqrt(385)*y*(13*z2 - 1)*(-10*x2*y2 + 5*x4 + y4)/(64*sqrt(pi))
    o[52] = (real_t)(4.1513246297620823f*xy*z*(x2 - y2)*(13.0f*z2 - 3.0f));                           // 3*sqrt(385)*xy*z*(x2 - y2)*(13*z2 - 3)/(8*sqrt(pi))
    o[53] = (real_t)(-0.15645893386229404f*y*(3.0f*x2 - y2)*(13.0f*z2*(11.0f*z2 - 3.0f) - 27.0f*z2 + 3.0f));                              // -3*sqrt(35)*y*(3*x2 - y2)*(13*z2*(11*z2 - 3) - 27*z2 + 3)/(64*sqrt(pi))
    o[54] = (real_t)(0.44253269244498261f*xy*z*(-110.0f*z2 + 143.0f*z4 + 15.0f));                              // 3*sqrt(70)*xy*z*(-110*z2 + 143*z4 + 15)/(32*sqrt(pi))
    o[55] = (real_t)(0.090331607582517306f*y*(-135.0f*z2 + 495.0f*z4 - 429.0f*z6 + 5.0f));                              // sqrt(105)*y*(-135*z2 + 495*z4 - 429*z6 + 5)/(64*sqrt(pi))
    o[56] = (real_t)(0.068284276912004949f*z*(315.0f*z2 - 693.0f*z4 + 429.0f*z6 - 35.0f));                              // sqrt(15)*z*(315*z2 - 693*z4 + 429*z6 - 35)/(32*sqrt(pi))
    o[57] = (real_t)(0.090331607582517306f*x*(-135.0f*z2 + 495.0f*z4 - 429.0f*z6 + 5.0f));                              // sqrt(105)*x*(-135*z2 + 495*z4 - 429*z6 + 5)/(64*sqrt(pi))
    o[58] = (real_t)(0.07375544874083044f*z*(x2 - y2)*(143.0f*z2*(3.0f*z2 - 1.0f) - 187.0f*z2 + 45.0f));                         // sqrt(70)*z*(x2 - y2)*(143*z2*(3*z2 - 1) - 187*z2 + 45)/(64*sqrt(pi))
    o[59] = (real_t)(-0.15645893386229404f*x*(x2 - 3.0f*y2)*(13.0f*z2*(11.0f*z2 - 3.0f) - 27.0f*z2 + 3.0f));                              // -3*sqrt(35)*x*(x2 - 3*y2)*(13*z2*(11*z2 - 3) - 27*z2 + 3)/(64*sqrt(pi))
    o[60] = (real_t)(1.0378311574405206f*z*(13.0f*z2 - 3.0f)*(-6.0f*x2*y2 + x4 + y4));                         // 3*sqrt(385)*z*(13*z2 - 3)*(-6*x2*y2 + x4 + y4)/(32*sqrt(pi))
    o[61] = (real_t)(-0.51891557872026028f*x*(13.0f*z2 - 1.0f)*(-10.0f*x2*y2 + x4 + 5.0f*y4));                          // -3*sqrt(385)*x*(13*z2 - 1)*(-10*x2*y2 + x4 + 5*y4)/(64*sqrt(pi))
    o[62] = (real_t)(2.6459606618019f*z*(15.0f*x2*y4 - 15.0f*x4*y2 + x6 - y6));                               // 3*sqrt(10010)*z*(15*x2*y4 - 15*x4*y2 + x6 - y6)/(64*sqrt(pi))
    o[63] = (real_t)(0.70716273252459627f*x*(-35.0f*x2*y4 + 21.0f*x4*y2 - x6 + 7.0f*y6));
}

// kernel
template <typename real_t>
__global__ void spherical_harmonics_encoding_kernel(
    std::uint32_t n,
    std::uint32_t degree,
    float const *xyz,
    real_t * const __restrict__ output
) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    #pragma unroll
    for (int i = index; i < n; i += stride) {
        real_t * const o = output + i * degree * degree;
        float x = xyz[i*3], y = xyz[i*3+1], z = xyz[i*3+2];

        sh_enc<real_t>(degree, x, y, z, o);
    }
}

template <typename real_t>
void spherical_harmonics_encoding_launcher(hipStream_t stream, void **buffers, char const *opaque, std::size_t opaque_len) {
    // inputs
    SphericalHarmonicsEncodingDescriptor const &desc =
        *deserialize<SphericalHarmonicsEncodingDescriptor>(opaque, opaque_len);
    std::uint32_t const n = desc.n;
    std::uint32_t const degree = desc.degree;
    float const *xyz = static_cast<float const *>(buffers[0]);  // [length, 3]

    // outputs
    real_t * const out = static_cast<real_t *>(buffers[1]);  // [length, dim * 2 * n_levels]

    int blockSize = 1024;
    int numBlocks = (n + blockSize - 1) / blockSize;
    numBlocks = std::min<int>(1024, numBlocks);
    spherical_harmonics_encoding_kernel<real_t><<<numBlocks, blockSize, 0, stream>>>(
            n,
            degree,
            xyz,
            out
        );
    check_throw(hipGetLastError());
}

}  // namespace

void spherical_harmonics_encoding_cuda_f32(
    hipStream_t stream,
    void **buffers,
    const char *opaque,
    std::size_t opaque_len
) {
    spherical_harmonics_encoding_launcher<float>(stream, buffers, opaque, opaque_len);
}

}  // namespace shjax
